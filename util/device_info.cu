#include "common_header.h"

hipError_t
get_device_version(int &device_version)
{
  int device;
  struct hipDeviceProp_t properties;
  hipError_t error;

  error = hipGetDevice (&device);
  if (error != hipSuccess) {
    fprintf (stdout,"failed to get device hipError_t=%d\n", error);
    return error;
  }
    
  error = hipGetDeviceProperties (&properties, device);
  if (error != hipSuccess) {
    fprintf (stdout,"failed to get properties hipError_t=%d\n", error);
    return error;
  } else {        
    device_version =  properties.major * 100 + properties.minor * 10;
  }
  return error;
}


