/* 
 * The MIT License (MIT)
 *
 * Copyright (c) 2020 NVIDIA
 *
 * Permission is hereby granted, free of charge, to any person obtaining a copy of
 * this software and associated documentation files (the "Software"), to deal in
 * the Software without restriction, including without limitation the rights to
 * use, copy, modify, merge, publish, distribute, sublicense, and/or sell copies of
 * the Software, and to permit persons to whom the Software is furnished to do so,
 * subject to the following conditions:
 * 
 * The above copyright notice and this permission notice shall be included in all
 * copies or substantial portions of the Software.
 * 
 * THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS OR
 * IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY, FITNESS
 * FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT. IN NO EVENT SHALL THE AUTHORS OR
 * COPYRIGHT HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER LIABILITY, WHETHER
 * IN AN ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING FROM, OUT OF OR IN
 * CONNECTION WITH THE SOFTWARE OR THE USE OR OTHER DEALINGS IN THE SOFTWARE.
 */
#include "common_header.h"

hipError_t
get_device_version(int &device_version)
{
  int device;
  struct hipDeviceProp_t properties;
  hipError_t error;

  error = hipGetDevice (&device);
  if (error != hipSuccess) {
    fprintf (stdout,"failed to get device hipError_t=%d\n", error);
    return error;
  }
    
  error = hipGetDeviceProperties (&properties, device);
  if (error != hipSuccess) {
    fprintf (stdout,"failed to get properties hipError_t=%d\n", error);
    return error;
  } else {        
    device_version =  properties.major * 100 + properties.minor * 10;
  }
  return error;
}


