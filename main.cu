#include "hip/hip_runtime.h"
/**
 * The MIT License (MIT)
 *
 * Copyright (c) 2020 NVIDIA
 *
 * Permission is hereby granted, free of charge, to any person obtaining a copy of
 * this software and associated documentation files (the "Software"), to deal in
 * the Software without restriction, including without limitation the rights to
 * use, copy, modify, merge, publish, distribute, sublicense, and/or sell copies of
 * the Software, and to permit persons to whom the Software is furnished to do so,
 * subject to the following conditions:
 * 
 * The above copyright notice and this permission notice shall be included in all
 * copies or substantial portions of the Software.
 * 
 * THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS OR
 * IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY, FITNESS
 * FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT. IN NO EVENT SHALL THE AUTHORS OR
 * COPYRIGHT HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER LIABILITY, WHETHER
 * IN AN ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING FROM, OUT OF OR IN
 * CONNECTION WITH THE SOFTWARE OR THE USE OR OTHER DEALINGS IN THE SOFTWARE.
 */
/******************************************************************************
 * Copyright (c) 2011-2020, NVIDIA CORPORATION.  All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are not permitted.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS" AND
 * ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE IMPLIED
 * WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE
 * DISCLAIMED. IN NO EVENT SHALL NVIDIA CORPORATION BE LIABLE FOR ANY
 * DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES
 * (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES;
 * LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND
 * ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE OF THIS
 * SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 *
 ******************************************************************************/

/* 4/18/2020 Derived from NVIDIA internal cublasMatmulBench, http://nvbugs/200591009
** Modified to create a GPU acceptance test utility on request from Microsoft
** http://nvbugs/2772765
** Purpose: Drive all GPU present to full power, TFLOPS and memory utilization and report PASS / FAIL
** Provide a watchdog timeout to detect hung tests.
** Test time is controlled by command line option T=<loop count>
** test timeout is hardcoded to 600 seconds per test.
** exit -1 on fail.
**
*/
#include "common_header.h"
#include "command_line.h"
#include "test_args.h"
#include "exceptions.h"
#include "memory.h"
#include "type_convert.h"
#include "common.h"
#include <hip/hip_runtime.h>

/* fault injection */
#include <thread>        
#include <chrono>         

/* watchdog includes; POSIX support on Windows with:
** 
https://docs.microsoft.com/en-us/cpp/build/vcpkg?view=vs-2019
https://github.com/microsoft/vcpkg.git
**
*/
#include <iostream>
#include <pthread.h>
#include <semaphore.h>
#include <stdio.h>
#include <stdlib.h>
#include <errno.h>
#include <time.h>

/* GST specific */
#include "GST.h"

/* Test metadata for watchdog oversight -------------------------------------------------------------- */
struct test_state {
    const char* test_name;
    int test_state; /* 1 is running, 0 is not running */
    time_t start_time;
    time_t end_time;
    int dummy;
};

/*Globals used by test meta data: TODO needs clean up:
** wrap in meta test class
*/
bool has_error = false;
bool test_ran = true;
struct test_state tstate[NUM_TESTS];
bool tests_done = false;
bool test_hung = false;
bool watchdog_bailed = false;

/*
 * Semaphores used for watchdog implementation:
 * wd - main() to watchdog to indicate test started
 * go - watchdog to main() to indicate run next test
 * done - main() to watchdog to indicate test complete 
 */
sem_t wd, go, done;

void reset_blas_opts(CommandLine& command_line, BlasOpts& blas_opts);

void* watchdog(void* in)
{
    printf("WATCHDOG starting, TIMEOUT: %d seconds\n", TEST_WAIT_TIME);

    int i = 0, n = 0;
    struct timespec ts;
    
    sem_post(&go);
    do {
        sem_wait(&wd);

        auto now = std::chrono::system_clock::now();
        auto secs = std::chrono::time_point_cast<std::chrono::seconds>(now);
        auto epoch_secs = secs.time_since_epoch();
        auto value_secs = std::chrono::duration_cast<std::chrono::seconds>(epoch_secs);
        ts.tv_sec = value_secs.count();
        ts.tv_nsec = 0L;
        ts.tv_sec += TEST_WAIT_TIME;
        n = sem_timedwait(&done, &ts);
        if ((n == -1) && (errno == ETIMEDOUT) && (tstate[i].test_state == 1)) {
            printf("TEST %s appears to be hung\n", tstate[i].test_name);
            printf("Terminating stress testing...\n");
            test_hung = true;
            sem_post(&go);
            break;
        }
        else if (n == -1) {
            perror("WATCHDOG sem_timedwait\n");
            printf("WATCHDOG thread exiting....\n");
            watchdog_bailed = true;
            pthread_exit(NULL);
        }
        sem_post(&go);
        i++;
    } while ((tests_done != true) || (test_hung != true));

    printf("WATCHDOG thread exiting....\n");
    pthread_exit(NULL);

    return(NULL);
}
/* ---------------------------------------------------------------------------------------------------------------------------*/

/*The base code for GST is cublasMatMulbench which accepts command 
**line arguments largely ignored by GST but left intact. Existing
** options include the time_loop "-T=<loop count>" which is used by GST
** and defaults to 100 requiring a runtime of around 30 min for five tests
** on a V100 for reference ad drives the GPU to full power, TFLOPS and memory
*/
using cublas::CommandLine;

/* From base code: cublasMatmulBench */
template <typename T_IN, typename T_OUT, typename T_MATH, typename T_SCALE>
static int 
lt_gemm(hipblasLtHandle_t ltHandle,
        const BlasOpts& blas_opts,
        T_IN *A,
        T_IN *B,
        T_OUT *C,
        T_SCALE alpha,
        T_SCALE beta,
        int lda,
        int ldb,
        int ldc) {
  try {
    hipblasLtMatmulDesc_t matmulDesc = NULL;
    const size_t workspaceSize = 1024 * 1024 * 4;
    void * workspace;
    int ldatransform = blas_opts.m_orderingA == HIPBLASLT_ORDER_COL ? lda : 32 * lda;
    int ldbtransform = 0;
    int ldctransform = blas_opts.m_orderingC == HIPBLASLT_ORDER_COL ? ldc : 32 * ldc; 

    switch(blas_opts.m_orderingB) {
      case CUBLASLT_ORDER_COL32_2R_4R4: // for ampere  
        ldbtransform = 32 * roundoff(ldb, 32);    
        break;
      case HIPBLASLT_ORDER_COL:
        ldbtransform = ldb;  
        break;
      default:
        ldbtransform = 32 * roundoff(ldb, 8);  
        break;
    }

    cublas::cuda_check_error(hipMalloc(&workspace, workspaceSize), "hipMalloc for workspace failed");

    hipblasLtMatrixLayout_t AtransformDesc = NULL, BtransformDesc = NULL, CtransformDesc = NULL;

   
    cublas::cublas_check_error(hipblasLtMatmulDescCreate(&matmulDesc, blas_opts.compute_type, blas_opts.scale_type),
                               "create MatmulDesc failed");
   

    /*
    cublas::cublas_check_error(hipblasLtMatmulDescCreate(&matmulDesc, HIPBLAS_COMPUTE_32I, blas_opts.scale_type),
        "create MatmulDesc failed");
    */

    cublas::cublas_check_error(hipblasLtMatmulDescSetAttribute(matmulDesc, HIPBLASLT_MATMUL_DESC_TRANSB, &blas_opts.transb, sizeof(blas_opts.transb)),
           "set DESC_TRANSB failed");
    cublas::cublas_check_error(hipblasLtMatmulDescSetAttribute(matmulDesc, HIPBLASLT_MATMUL_DESC_TRANSA, &blas_opts.transa, sizeof(blas_opts.transa)),
           "set DESC_TRANSA failed");
    cublas::cublas_check_error(hipblasLtMatmulDescSetAttribute(matmulDesc, CUBLASLT_MATMUL_DESC_TRANSC, &blas_opts.transc, sizeof(blas_opts.transc)),
           "set DESC_TRANSC failed");


    // ---------------------------------------------------------------------------------------------
    // create descriptors for transformed matrices

    cublas::cublas_check_error(hipblasLtMatrixLayoutCreate(&AtransformDesc, blas_opts.input_type, 
           blas_opts.transa == HIPBLAS_OP_N ? blas_opts.m : blas_opts.k, blas_opts.transa == HIPBLAS_OP_N ? blas_opts.k : blas_opts.m, ldatransform),
           "create MatrixLayout for AtransformDesc failed");
    cublas::cublas_check_error(hipblasLtMatrixLayoutSetAttribute(
            AtransformDesc, HIPBLASLT_MATRIX_LAYOUT_ORDER, &blas_opts.m_orderingA, sizeof(blas_opts.m_orderingA)),
            "set LAYOUT_ORDER for AtransformDesc failed");
    cublas::cublas_check_error(hipblasLtMatrixLayoutCreate(&BtransformDesc, blas_opts.input_type, 
            blas_opts.transb == HIPBLAS_OP_N ? blas_opts.k : blas_opts.n, blas_opts.transb == HIPBLAS_OP_N ? blas_opts.n : blas_opts.k, ldbtransform),
           "create MatrixLayout for BtransformDesc failed");
    cublas::cublas_check_error(hipblasLtMatrixLayoutSetAttribute(
            BtransformDesc, HIPBLASLT_MATRIX_LAYOUT_ORDER, &blas_opts.m_orderingB, sizeof(blas_opts.m_orderingB)),
            "set LAYOUT_ORDER for BtransformDesc failed");

    cublas::cublas_check_error(hipblasLtMatrixLayoutCreate(&CtransformDesc, blas_opts.output_type, blas_opts.m, blas_opts.n, ldctransform),
           "create MatrixLayout for CtransformDesc failed");
    cublas::cublas_check_error(hipblasLtMatrixLayoutSetAttribute(
           CtransformDesc, HIPBLASLT_MATRIX_LAYOUT_ORDER, &blas_opts.m_orderingC, sizeof(blas_opts.m_orderingC)),
           "set LAYOUT_ORDER for CtransformDesc failed");

    // ---------------------------------------------------------------------------------------------
    // computation
    char ta = operation_to_char(blas_opts.transa);
    char tb = operation_to_char(blas_opts.transb);

    printf("#### args: ta=%c tb=%c m=%d n=%d k=%d", ta, tb, blas_opts.m, blas_opts.n, blas_opts.k);
    printf(" lda=%d ldb=%d ldc=%d loop=%d\n", ldatransform, ldbtransform, ldctransform, blas_opts.timing_loop);   

    /*
    printf ("#### args: ta=%c tb=%c m=%d n=%d k=%d", ta, tb, blas_opts.m, blas_opts.n, blas_opts.k);
    printCuType( " alpha =", alpha);
    printCuType( " beta=", beta);
    printf("\n");
    printf("#### args: lda=%d ldb=%d ldc=%d loop=%d\n", ldatransform, ldbtransform, ldctransform, blas_opts.timing_loop);   
    printf("#### input_type %d output_type %d scale_type %d math_type %d compute_type %d\n",
        blas_opts.input_type, blas_opts.output_type, blas_opts.scale_type, blas_opts.math_type, blas_opts.compute_type);
    */
    using namespace std::chrono;
    high_resolution_clock::time_point start = high_resolution_clock::now();

    for (int i = 0; i < blas_opts.timing_loop; ++i) {
        cublas::cublas_check_error(hipblasLtMatmul(ltHandle,
                                                matmulDesc,
                                                &alpha,
                                                A,
                                                AtransformDesc,
                                                B,
                                                BtransformDesc,
                                                &beta,
                                                C,
                                                CtransformDesc,
                                                C,
                                                CtransformDesc,
                                                NULL,
                                                workspace,
                                                workspaceSize,
                                                0),"hipblasLtMatmul failed");    
    }

    cublas::cuda_check_error(hipDeviceSynchronize(), "hipDeviceSynchronize failed");



    high_resolution_clock::time_point end = high_resolution_clock::now();
    duration<double> time_span = duration_cast<duration<double>>(end - start);
    double flopsCoef = 2.0;

    if ((blas_opts.math_type == HIP_C_32F) || (blas_opts.math_type == HIP_C_64F)) {
      flopsCoef = 8.0;
    }

    double TheoreticalFlops = flopsCoef * (double)blas_opts.m * (double)blas_opts.n * (double)blas_opts.k;    
    double  TheoreticalBandwidth =
                sizeof(T_IN) * ((double)blas_opts.m * (double)blas_opts.k + (double)blas_opts.k * (double)blas_opts.n) +
                sizeof(T_OUT) * (double)blas_opts.m * (double)blas_opts.n;    
    /*
    if (blas_opts.timing_only) {
      fprintf (stdout, "!!!! GPU timing only. CPU reference not run.\n");
      double cudaGflops = blas_opts.timing_loop * (1e-9*TheoreticalFlops)/(time_span.count());
      double cudaBandwidth = blas_opts.timing_loop *(1e-9*TheoreticalBandwidth)/(time_span.count());
      cublasPrintPerf(false, time_span.count(), cudaGflops);//, cudaBandwidth );    
      printGemmSOL<T_MATH>(0, time_span.count(), blas_opts.timing_loop, blas_opts.m, blas_opts.n, blas_opts.k, (int)(blas_opts.algo));
    }
    */

    // descriptors are no longer needed as all GPU work was already enqueued
    if (CtransformDesc) cublas::cublas_check_error(hipblasLtMatrixLayoutDestroy(CtransformDesc), "destory CtransformDesc failed");
    if (BtransformDesc) cublas::cublas_check_error(hipblasLtMatrixLayoutDestroy(BtransformDesc), "destory BtransformDesc failed");
    if (AtransformDesc) cublas::cublas_check_error(hipblasLtMatrixLayoutDestroy(AtransformDesc), "destory AtransformDesc failed");
    if (matmulDesc) cublas::cublas_check_error(hipblasLtMatmulDescDestroy(matmulDesc), "destroy matmulDesc failed");
    if (workspace) cublas::cuda_check_error(hipFree(workspace), "free workspace failed");    
  } catch (cublas::cuda_exception &e) {
    cout << e << endl;  
    return 1;
  } catch (cublas::cublas_exception &e) {
    cout << e << endl;  
    return 1;
  } catch (const std::exception & e){
    cout << e.what() << endl;  
    return 1;  
  }
 
  return 0;
}

template <typename T_IN, typename T_OUT, typename T_MATH, typename T_SCALE>
static void
test_engine(const BlasOpts& blas_opts) {
  /* printf("testing cublasLt\n"); */
  try {
    T_IN *d_A = nullptr;
    T_IN *d_B = nullptr;      
    T_OUT *d_C = nullptr;
    T_SCALE alpha = cuGet<T_SCALE>(blas_opts.alpha);
    T_SCALE beta = cuGet<T_SCALE>(blas_opts.beta);
    int matrixM = 0, matrixN = 0, matrixK = 0;
    int rowsA = 0, rowsB = 0, rowsC = 0;
    int colsA = 0, colsB = 0, colsC = 0;
    size_t matrixSizeA = 0, matrixSizeB = 0, matrixSizeC = 0;    

    matrixM = blas_opts.m;
    matrixN = blas_opts.n;
    matrixK = blas_opts.k;
    
    if (blas_opts.lda)  {
      if ((blas_opts.transa == HIPBLAS_OP_N) && (blas_opts.lda < matrixM)) {
          fprintf (stdout, "lda(=%d) must be bigger than m(=%d)\n", blas_opts.lda , matrixM);
          return;
      }
      if ((blas_opts.transa != HIPBLAS_OP_N) && (blas_opts.lda < matrixK)) {
          fprintf (stdout, "lda(=%d) must be bigger than k(=%d) for ta\n", blas_opts.lda , matrixK);
          return;
      }
    }
    if (blas_opts.ldb)  {
      if ((blas_opts.transb == HIPBLAS_OP_N) && (blas_opts.ldb < matrixK)) {
          fprintf (stdout, "ldb(=%d) must be bigger than k(=%d)\n", blas_opts.ldb , matrixK);
          return;
      }
      if ((blas_opts.transb != HIPBLAS_OP_N) && (blas_opts.ldb < matrixN)) {
          fprintf (stdout, "ldb(=%d) must be bigger than n(=%d) for tb\n", blas_opts.ldb , matrixN);
          return;
      }

    }
    if ((blas_opts.ldc) && (blas_opts.ldc < matrixM)) {
      fprintf (stdout, "ldc(=%d) must be bigger than m(=%d)\n", blas_opts.ldc , matrixM);
      return;
    }    

    if (blas_opts.transa != HIPBLAS_OP_N) {
      rowsA = imax(blas_opts.lda, matrixK);
      colsA = matrixM;
    } else {
      rowsA = imax(blas_opts.lda, matrixM);  
      colsA = matrixK;
    }
    if (blas_opts.transb != HIPBLAS_OP_N) {
      rowsB = imax(blas_opts.ldb, matrixN);
      colsB = matrixK;
    } else {
      rowsB = imax(blas_opts.ldb, matrixK);  
      colsB = matrixN;
    }
    rowsC = imax(blas_opts.ldc, matrixM);
    colsC = matrixN;        

    matrixSizeA = (size_t)rowsA * colsA;
    matrixSizeB = (size_t)rowsB * colsB;
    matrixSizeC = (size_t)rowsC * colsC;

    printf("#### args: matrixSizeA %lld matrixSizeB %lld matrixSizeC %lld \n", matrixSizeA, matrixSizeB, matrixSizeC);


// ----------------- debug below 
#ifdef DEBUG_MATRIX_SIZES
printf("***** TEST PASSED ****\n");
return;
#endif
// ------------------ debug above 

    d_A = cublas::device_memory::allocate<T_IN>(matrixSizeA);
    d_B = cublas::device_memory::allocate<T_IN>(matrixSizeB);
    d_C = cublas::device_memory::allocate<T_OUT>(matrixSizeC);
    

    //printf("DEBUG: After  cublas::device_memory::allocate\n");

    //cublas::cuda_check_error(hipMemset(d_C, 0, matrixSizeC * sizeof(h_C[0])), "hipMemset error");
    
    hipblasLtHandle_t ltHandle;
    cublas::cublas_check_error(hipblasLtCreate(&ltHandle), "create cublasLt handle failed");
    

    if(lt_gemm<T_IN, T_OUT, T_MATH, T_SCALE>(ltHandle, blas_opts, d_A, d_B, d_C, alpha, beta, rowsA, rowsB, rowsC)) {
      has_error = true;  
    }

    test_ran = true;

    cublas::device_memory::free(d_A);
    cublas::device_memory::free(d_B);
    cublas::device_memory::free(d_C);
    cublas::cublas_check_error(hipblasLtDestroy(ltHandle), "destroy ltHandle failed");

/*
    if (has_error) {
      printf("testing cublasLt fail\n");    
      exit(-1);
    } else {
      printf("testing cublasLt pass\n");
    }
*/

  } catch (cublas::cuda_exception &e) {
    cout << e << endl;  
    printf("testing cublasLt fail\n");
    exit(-1);
  } catch (cublas::cublas_exception &e) {
    cout << e << endl;
    printf("testing cublasLt fail\n");
    exit(-1);
  } catch (const std::exception & e){
    cout << e.what() << endl;
    printf("testing cublasLt fail\n");
    exit(-1);
  }

}

static void
test_cublasLt(BlasOpts& blas_opts) {
  try{    

    printf("#### math_type %d  \n", blas_opts.math_type );

    switch(blas_opts.math_type) {
      case HIP_R_32F: //sss A,B : FP32 ->  C FP32
        if ((blas_opts.input_type == HIP_R_32F) &&
            (blas_opts.output_type == HIP_R_32F) &&
            (blas_opts.scale_type == HIP_R_32F)) {
#ifdef DEBUG_MATRIX_SIZES
printf("%s\n", "test_engine<float, float, float, float>(blas_opts)");
#endif
          test_engine<float, float, float, float>(blas_opts);
        } //hss A,B FP16 ->  C FP32 
        if ((blas_opts.input_type == HIP_R_16F) &&
            (blas_opts.output_type == HIP_R_32F) &&
            (blas_opts.scale_type == HIP_R_32F)) {
#ifdef DEBUG_MATRIX_SIZES
printf("%s\n", "test_engine<__half, float, float, float>(blas_opts)");
#endif
          test_engine<__half, float, float, float>(blas_opts);
        } // hsh A,B FP16 ->  C FP16
        if ((blas_opts.input_type == HIP_R_16F) &&
            (blas_opts.output_type == HIP_R_16F) &&
            (blas_opts.scale_type == HIP_R_32F)) {
#ifdef DEBUG_MATRIX_SIZES
printf("%s\n", "test_engine<__half, __half, float, float>(blas_opts)");
#endif
          test_engine<__half, __half, float, float>(blas_opts);
        } 
        break;
      case HIP_C_32F: //ccc
        if ((blas_opts.input_type == HIP_C_32F) &&
            (blas_opts.output_type == HIP_C_32F) &&
            (blas_opts.scale_type == HIP_C_32F)) {
#ifdef DEBUG_MATRIX_SIZES
printf("%s\n", "test_engine<hipComplex, hipComplex, hipComplex, hipComplex>(blas_opts)");
#endif
          test_engine<hipComplex, hipComplex, hipComplex, hipComplex>(blas_opts);
        } 
        break; 
      case HIP_R_64F: //ddd A,B : FP64 ->  C FP64
        if ((blas_opts.input_type == HIP_R_64F) &&
            (blas_opts.output_type == HIP_R_64F) &&
            (blas_opts.scale_type == HIP_R_64F)) {
#ifdef DEBUG_MATRIX_SIZES
printf("%s\n", "test_engine<double, double, double, double>(blas_opts)");
#endif
          test_engine<double, double, double, double>(blas_opts);
        } 
        break;
      case HIP_C_64F: // zzz 
          if ((blas_opts.input_type == HIP_C_64F) &&
              (blas_opts.output_type == HIP_C_64F) &&
              (blas_opts.scale_type == HIP_C_64F)) {
#ifdef DEBUG_MATRIX_SIZES
printf("%s\n", "test_engine<hipDoubleComplex, hipDoubleComplex, hipDoubleComplex, hipDoubleComplex>(blas_opts)");
#endif
              test_engine<hipDoubleComplex, hipDoubleComplex, hipDoubleComplex, hipDoubleComplex>(blas_opts);
              /* DEBUG: test watchdog timeout detection and error exit by uncommenting to inject a timeout error
              std::this_thread::sleep_for(std::chrono::seconds(600));
              */
          }
        break;
      case HIP_R_16F: // hhh   
        if ((blas_opts.input_type == HIP_R_16F) &&
            (blas_opts.output_type == HIP_R_16F) &&
            (blas_opts.scale_type == HIP_R_16F)) {
#ifdef DEBUG_MATRIX_SIZES
printf("%s\n", "test_engine<__half, __half, __half,__half>(blas_opts)");
#endif
          test_engine<__half, __half, __half,__half>(blas_opts);
        } 
        break;
      case HIP_R_32I: {//bisb_imma
          int device_version = 0;
          cublas::cuda_check_error(get_device_version(device_version), "get device version failed");          
// --------------- debug below 
#ifndef DEBUG_MATRIX_SIZES
          if (device_version < 750) {
            printf("not supported for the imma options\n");
	        test_ran = false;
            return;
          }
#endif
// --------------- debug above
          blas_opts.m_orderingA = CUBLASLT_ORDER_COL32;
          blas_opts.m_orderingB = device_version >= 800 ? CUBLASLT_ORDER_COL32_2R_4R4 : CUBLASLT_ORDER_COL4_4R2_8C;
          blas_opts.m_orderingC = CUBLASLT_ORDER_COL32;
          blas_opts.transa = HIPBLAS_OP_N;
          blas_opts.transb = HIPBLAS_OP_T; 
          if ((blas_opts.input_type == HIP_R_8I) &&
              (blas_opts.output_type == HIP_R_8I) &&
              (blas_opts.scale_type == HIP_R_32F)) {
              
#ifdef DEBUG_MATRIX_SIZES
printf("%s\n", "test_engine<int8_t, int8_t, int, float>(blas_opts)");
#endif
            test_engine<int8_t, int8_t, int, float>(blas_opts);
          } //bii_imma
          if ((blas_opts.input_type == HIP_R_8I) &&
              (blas_opts.output_type == HIP_R_32I) &&
              (blas_opts.scale_type == HIP_R_32I)) {
#ifdef DEBUG_MATRIX_SIZES
printf("%s\n", "test_engine<int8_t, int, int, int>(blas_opts)");
#endif
            test_engine<int8_t, int, int, int>(blas_opts);
          } 
        }
        break;
      default:
        printf("mode not supported\n");
        break;
    }    
  } catch (cublas::cuda_exception &e) {
    cout << e << endl;  
    printf("testing cublasLt fail\n");
    exit(-1);
  } catch (const std::exception & e){
    cout << e.what() << endl;
    printf("testing cublasLt fail\n");
    exit(-1);
  }    
}

/* ------------------------------------------------------------------------------------------------------------------------------- */

int main(int argc, char *argv[]) {
  int ret = 0;
  pthread_t wd_thread;
  pthread_attr_t attr;
  GST gst;

  sem_init(&wd, 0, 0);
  sem_init(&go, 0, 0);
  sem_init(&done, 0, 0);
  void(*watchdog(void*));

  if (pthread_attr_init(&attr)) {
    perror("pthread_attr_init - watchdog");
    exit(-1);
  }

  if (pthread_create(&wd_thread, &attr, watchdog, (void*)NULL) != 0) {
      perror("pthread create - watchdog");
      exit(-1);
  }

  printf("%s capturing GPU information...\n", argv[0]);

  int deviceCount = 0;
  hipGetDeviceCount(&deviceCount);

  if (deviceCount == 0) {
        printf("There are no available device(s) that support CUDA\n");
	printf("Exiting...\n");
	exit(-1);
  } else {
        printf("Detected %d CUDA Capable device(s)\n", deviceCount);
  }

  CommandLine command_line(argc, argv);

  int device_arg = -1;
  if (command_line.check_cmd_line_flag("dv")) {
    command_line.get_cmd_line_argument("dv", device_arg);
    // arg check 1
    if (device_arg > deviceCount) {
      printf("Device (dv) #%d parameter is too big\n", device_arg);
      exit(1);
    }
    // arg check 2
    if (device_arg < 0) {
      printf("Device (dv) #%d parameter is too small\n", device_arg);
      exit(1);
    }
    printf("Device #%d is selected\n", device_arg);
  }

  BlasOpts blas_opts;
  parse_args(command_line, blas_opts);
  reset_blas_opts(command_line, blas_opts);

  /* GPU detection and test initilization */
  int dev;
  size_t gpumem = 0LL;
  hipDeviceProp_t devprops[MAX_NUM_GPUS] {};
  for (dev = 0; dev < deviceCount; dev++) {
      if ((device_arg >= 0)  && (device_arg != dev)) {
        printf("Device %d: skiped\n", dev);
        continue;
      }
      CHECK(hipSetDevice(dev));
      CHECK(hipGetDeviceProperties(&devprops[dev], dev));
      printf("Device %d: \"%s\"\n", dev, devprops[dev].name);
      if (dev == 0)
          gpumem = devprops[dev].totalGlobalMem;
      else {
          if (gpumem != devprops[dev - 1].totalGlobalMem) {
              printf("Detected different GPU memory sizes\n");
              printf("gpumem: %lld, GPU %d %lld\n", (long long) gpumem, (dev - 1), (long long) devprops[dev - 1].totalGlobalMem);
              printf("EXITING...\n");
              exit(0);
          }
      }
  }
  gpumem /= (1024 * 1024 * 1024);

  /* Initilize tests based on type of GPU
  */
  int memgb = 0;

#ifndef DEBUG_MATRIX_SIZES
  string gpu_name(devprops[0].name);
#else
// ------------------------ debug below
// These entries should match GST::test_suite; clever C++ way to range over the enum and cast to string not obvious...
for (string gpu_name :  {"T4", "A100_40", "A100_80", "K80", "M60", "P40", "P100", "H100", "V100_16", "V100_32", "Generic"}) {

if (!gpu_name.compare(string("A100_80"))) { 
        printf("set A100_80\n");
        gpumem = 80;
}
else if (!gpu_name.compare(string("V100_32"))) {
        printf("set V100_32\n");
        gpumem = 32;
}


printf("DEBUG_MATRIX_SIZES: Checking matrix size only (no CUDA execution) for: %s\n", gpu_name.c_str());
#endif
// ------------------------ debug above
  
  while (true) {
    if (gpu_name.find("A100", 0) != string::npos) {

        if (gpumem > 40) {
          cout << "Initilizing A100 80 GB based test suite" << endl;
          gst = GST(GST::A100_80);
          memgb = 80;
        }  else {
          cout << "Initilizing A100 40 GB based test suite" << endl;
          gst = GST(GST::A100_40);
          memgb = 40;
        }
        break;
    }
    if (gpu_name.find("T4", 0) != string::npos) {
        cout << "Initilizing T4 based test suite" << endl;
        gst = GST(GST::T4);
        memgb = 16;
        break;
    }
    if (gpu_name.find("K80", 0) != string::npos) {
        cout << "Initilizing K80 based test suite" << endl;
        gst = GST(GST::K80);
        memgb = 11;
        break;
    }
    if (gpu_name.find("M60", 0) != string::npos) {
        cout << "Initilizing M60 based test suite" << endl;
        gst = GST(GST::M60);
        memgb = 8;
        break;
    }
    if (gpu_name.find("P40", 0) != string::npos) {
        cout << "Initilizing P40 based test suite" << endl;
        gst = GST(GST::P40);
        memgb = 22;
        break;
    }
    if (gpu_name.find("P100", 0) != string::npos) {
        cout << "Initilizing P100 based test suite" << endl;
        gst = GST(GST::P100);
        memgb = 16;
        break;
    }
    if (gpu_name.find("V100", 0) != string::npos) {

        if (gpumem > 30) {
          cout << "Initilizing V100 32 GB based test suite" << endl;
          gst = GST(GST::V100_32);
          memgb = 32;
        }  else {
          cout << "Initilizing V100 16 GB based test suite" << endl;
          gst = GST(GST::V100_16);
          memgb = 16;
        }
        break;
    }
    if (gpu_name.find("H100", 0) != string::npos) {
        cout << "Initilizing H100 based test suite" << endl;
        gst = GST(GST::H100);
        memgb = 95;
        break;
    }
    cout << "Initilizing Generic test suite" << endl;
    gst = GST(GST::Generic);
    memgb = 8;
    break;
  }


  printf("GPU Memory: %lld, memgb: %d\n", (long long) gpumem, memgb);
  printf("\n\n");


  for (dev = 0; dev < deviceCount; dev++) {
	CHECK(hipSetDevice(dev));
	printf("Device %d: \"%s\", PCIe: %x\n", dev, devprops[dev].name,devprops[dev].pciBusID);

       // gst.dump_test_args(0);

	for (int t_num = 0; t_num  < NUM_TESTS; t_num++) {

            /* Abort if watchdog has died */
            if (watchdog_bailed) {
                printf("WATCHDOG Thread exited...\n");
                printf("GPUstress terminating\n");
                exit(-1);
            }
            reset_blas_opts(command_line, blas_opts);
            /* Debug
            gst.dump_test_args(tix);
            hello_world(blas_opts, gst.stress_tests[0].P_arg);
            */

            /* Parse command line optioms */
            bool p_parse = parse_in_math_scale_out_type(blas_opts, gst.stress_tests[t_num].P_arg);
            // cout << "DEBUG:" << "after parse" << endl;
            if (!p_parse) {
                printf("p_parse failed\n");
                exit(-1);
            }
            // cout << "DEBUG:" << "set opts" << endl;

            blas_opts.m = gst.stress_tests[t_num].m_arg;
            blas_opts.n = gst.stress_tests[t_num].n_arg;
            blas_opts.k = gst.stress_tests[t_num].k_arg;
            blas_opts.m_opt = true;
            blas_opts.n_opt = true;
            blas_opts.k_opt = true;

            if (gst.stress_tests[t_num].ta_arg == 1)
                blas_opts.transa_opt = true;
            else {
                blas_opts.transa_opt = false;
                blas_opts.transa = (hipblasOperation_t)0;
            }
            if (gst.stress_tests[t_num].tb_arg == 1)
                blas_opts.transb_opt = true;
            else {
                blas_opts.transb_opt = false;
                blas_opts.transb = (hipblasOperation_t)0;
            }
            blas_opts.beta = 0.0f;
            blas_opts.beta_opt = true;
        
            printf("\n***** STARTING TEST %d: %s On Device %d %s\n", t_num, gst.stress_tests[t_num].test_name, dev, devprops[dev].name);
            fflush(stdout);
            tstate[t_num].test_name = gst.stress_tests[t_num].test_name;
            tstate[t_num].test_state = 1;
            tstate[t_num].start_time = time(NULL);
            // cout << "DEBUG:" << "signal wd" << endl;
        
            /* Signal watchdog test started */
            sem_post(&wd);
            // cout << "DEBUG:" << "start test" << endl;

            /* Run the test */
            test_cublasLt(blas_opts);
            printf("***** TEST %s On Device %d %s\n", gst.stress_tests[t_num].test_name, dev, devprops[dev].name);

            if (!test_ran) 
                printf("***** TEST DID NOT EXECUTE *****\n\n");
            else {
                if (has_error == true || test_hung == true) {
                    printf("***** TEST FAILED ****\n\n");
                    ret = -1;
                    break;
                }
            else
#ifndef DEBUG_MATRIX_SIZES
// ----------------------- debug below
                printf("***** TEST PASSED ****\n");
// ----------------------- debug above (commend out)
#endif
              continue;
            }
            tstate[t_num].end_time = time(NULL);
            tstate[t_num].test_state = 0;
            printf("TEST TIME: %d seconds\n",(int)(tstate[t_num].end_time - tstate[t_num].start_time));
            hipDeviceSynchronize();
            hipDeviceReset();

            if (t_num == NUM_TESTS)
                tests_done = true;

            /* Signal watchdog test finished*/
            sem_post(&done);

            /* wait for watchdog to signal next next test */
            sem_wait(&go);
      }
  }
//------------------- debug below
#ifdef DEBUG_MATRIX_SIZES
    gpumem=0;
}
#endif
// ----------------- debug above
  
  exit(ret);
}

